#include "hip/hip_runtime.h"
// output example
/*
result[0] = 16.81, result[1] = 16.81, result[2] = 16.81, result[3] = 16.81, result[4] = 16.81, 
result[4194299] = 16.81, result[4194300] = 16.81, result[4194301] = 16.81, result[4194302] = 16.81, result[4194303] = 16.81, 
Success! All values calculated correctly.
*/

#include <stdio.h>

__global__
void initWith(double num, double *vector, int N)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    for(int i = index; i < N; i += stride)
    {
        vector[i] = num;
    }
}

__global__
void saxpy(double *a, double *b, double *result, int N)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    for(int i = index; i < N; i += stride)
        result[i] = 2 * a[i] + b[i];
}

// first check
void checkFiveElements(double *result, int N)
{    
    // Print out the first and last 5 values of result for a quality check
    for( int i = 0; i < 5; ++i )
        printf("result[%d] = %.2f, ", i, result[i]);
    printf ("\n");
    
    for( int i = N - 5; i < N; ++i )
        printf("result[%d] = %.2f, ", i, result[i]);
    printf ("\n");
}

// second check
void checkAllElements(double target, double *result, int N)
{    
    for(int i = 0; i < N; ++i)
    {
        if(result[i] != target)
        {
            printf("FAIL: vector[%d] - %0.0f does not equal %0.0f\n", i, result[i], target);
            exit(1);
        }
    }
    
    printf("Success! All values calculated correctly.\n");
}

int main()
{
    double *a, *b, *result;
    int deviceId;
    int numberOfSMs;

    // get number of Sms
    hipGetDevice(&deviceId);
    hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);

    int N = 2048 * 2048; // Number of elements in each vector
    size_t size = N * sizeof(double); // The total number of bytes per vector

    // Allocate memory 
    hipMallocManaged(&a, size);
    hipMallocManaged(&b, size);
    hipMallocManaged(&result, size);

    hipMemPrefetchAsync(a, size, deviceId);
    hipMemPrefetchAsync(b, size, deviceId);
    hipMemPrefetchAsync(result, size, deviceId);

    // Number of threads and blocks
    size_t threadsPerBlock = 256;
    size_t numberOfBlocks = 32 * numberOfSMs;
    
    // For error handling
    hipError_t saxpyErr;
    hipError_t asyncErr;

    // Initialize vectors
    initWith<<<numberOfBlocks, threadsPerBlock>>>(4.84, a, N);
    initWith<<<numberOfBlocks, threadsPerBlock>>>(7.13, b, N);
    initWith<<<numberOfBlocks, threadsPerBlock>>>(0, result, N);

    saxpy<<<numberOfBlocks, threadsPerBlock>>>(a, b, result, N);

    // handle errors
    saxpyErr = hipGetLastError();
    if(saxpyErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(saxpyErr));
    
    // Wait for the GPU to finish
    asyncErr = hipDeviceSynchronize(); 
    if(asyncErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(asyncErr));
    
    hipMemPrefetchAsync(result, size, hipCpuDeviceId);
    checkFiveElements(result, N);
    checkAllElements(2 * a[0] + b[0], result, N);

    // Free all our allocated memory
    hipFree(a);
    hipFree(b);
    hipFree(result);
}
